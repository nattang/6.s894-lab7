#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

////////////////////////////////////////////////////////////////////////////////
// Utility Functions

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

void rle_compress_cpu(
    uint32_t raw_count,
    char const *raw,
    std::vector<char> &compressed_data,
    std::vector<uint32_t> &compressed_lengths) {
    compressed_data.clear();
    compressed_lengths.clear();

    uint32_t i = 0;
    while (i < raw_count) {
        char c = raw[i];
        uint32_t run_length = 1;
        i++;
        while (i < raw_count && raw[i] == c) {
            run_length++;
            i++;
        }
        compressed_data.push_back(c);
        compressed_lengths.push_back(run_length);
    }
}

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// Optimized GPU Implementation

namespace rle_gpu {

// Reduce each block & store into workspace[blockIdx.x]
template <typename Op>
__global__ void upstream_scan(
    size_t n,
    typename Op::Data *x,        // pointer to GPU memory
    typename Op::Data *workspace // pointer to GPU memory
) {
    using Data = typename Op::Data;

    extern __shared__ __align__(16) char shmem_raw[]; // OK
    Data *shmem = reinterpret_cast<Data *>(shmem_raw);

    int threadId = threadIdx.x;
    int threads_per_block = blockDim.x;
    int block_offset = blockIdx.x * threads_per_block * VALS_PER_THREAD;

    // load from global memory & perform thread scan
    Data vals[VALS_PER_THREAD];
    for (int i = 0; i < VALS_PER_THREAD; i++) {
        int idx = block_offset + threadId * VALS_PER_THREAD + i;
        if (idx >= n) {
            vals[i] = Op::identity();
        } else {
            vals[i] = x[idx];
        }
    }
    for (int i = 1; i < VALS_PER_THREAD; i++) {
        vals[i] = Op::combine(vals[i - 1], vals[i]);
    }

    Data thread_sum = vals[VALS_PER_THREAD - 1];
    shmem[threadId] = thread_sum;

    // scan across shmem (across all warps in the block)
    for (int i = 1; i < threads_per_block; i <<= 1) {
        __syncthreads();
        Data cur_val = shmem[threadId];
        if (threadId >= i) {
            cur_val = Op::combine(shmem[threadId - i], cur_val);
        }
        __syncthreads();
        shmem[threadId] = cur_val;
    }
    __syncthreads();

    // add prev warp reduction to each thread
    Data threadPrefix = Op::identity();
    if (threadId > 0) { // mask first warp
        threadPrefix = shmem[threadId - 1];
    }
    for (int i = 0; i < VALS_PER_THREAD; i++) {
        vals[i] = Op::combine(threadPrefix, vals[i]);
    }

    // write back to x
    for (int i = 0; i < VALS_PER_THREAD; i++) {
        int idx = block_offset + threadId * VALS_PER_THREAD + i;
        if (idx >= n) {
            break;
        }
        x[idx] = vals[i];
    }
    // write blockSum to workspace
    if (threadId == threads_per_block - 1) {
        workspace[blockIdx.x] = shmem[threads_per_block - 1];
    }
}

template <typename Op>
__global__ void spine_scan(
    // size_t vals_per_thread,
    typename Op::Data *blocksums // pointer to GPU memory
) {
    using Data = typename Op::Data;

    extern __shared__ __align__(16) char shmem_raw[]; // OK
    Data *shmem = reinterpret_cast<Data *>(shmem_raw);

    int threads_per_block = blockDim.x;
    int threadId = threadIdx.x;

    Data vals[SPINE_VALS_PER_THREAD];
    for (int i = 0; i < SPINE_VALS_PER_THREAD; i++) {
        int idx = threadId * SPINE_VALS_PER_THREAD + i;
        vals[i] = blocksums[idx];
    }

    for (int i = 1; i < SPINE_VALS_PER_THREAD; i++) {
        vals[i] = Op::combine(vals[i - 1], vals[i]);
    }

    Data thread_sum = vals[SPINE_VALS_PER_THREAD - 1];
    shmem[threadId] = thread_sum;

    // scan across shmem
    for (int i = 1; i < threads_per_block; i <<= 1) {
        __syncthreads();
        Data cur_val = shmem[threadId];
        if (threadId >= i) {
            cur_val = Op::combine(shmem[threadId - i], cur_val);
        }
        __syncthreads();
        shmem[threadId] = cur_val;
    }
    __syncthreads();

    Data threadPrefix = Op::identity();
    if (threadId > 0) {
        threadPrefix = shmem[threadId - 1];
    }

    for (int i = 0; i < SPINE_VALS_PER_THREAD; i++) {
        vals[i] = Op::combine(threadPrefix, vals[i]);
    }

    for (int i = 0; i < SPINE_VALS_PER_THREAD; i++) {
        int idx = threadId * SPINE_VALS_PER_THREAD + i;
        blocksums[idx] = vals[i];
    }
}

template <typename Op>
__global__ void downstream_scan_fix(
    typename Op::Data *x,        // pointer to GPU memory
    typename Op::Data *blocksums // pointer to GPU memory
) {
    using Data = typename Op::Data;

    int threads_per_block = blockDim.x;
    int block_offset = blockIdx.x * threads_per_block * VALS_PER_THREAD;

    int threadId = threadIdx.x;

    Data vals[VALS_PER_THREAD];
    for (int i = 0; i < VALS_PER_THREAD; i++) {
        int idx = block_offset + threadId * VALS_PER_THREAD + i;
        vals[i] = x[idx];
    }

    Data block_prefix = Op::identity();
    if (blockIdx.x > 0) {
        block_prefix = blocksums[blockIdx.x - 1];
    }

    for (int i = 0; i < VALS_PER_THREAD; i++) {
        int idx = block_offset + threadId * VALS_PER_THREAD + i;
        vals[i] = Op::combine(block_prefix, vals[i]);
    }

    for (int i = 0; i < VALS_PER_THREAD; i++) {
        int idx = block_offset + threadId * VALS_PER_THREAD + i;
        x[idx] = vals[i];
    }
}

// Returns desired size of scratch buffer in bytes.
template <typename Op> size_t get_workspace_size(size_t n) {
    int num_blocks = CEIL_DIV(n, WARPS_PER_BLOCK * 32);
    return num_blocks * sizeof(uint32_t) * 2; // double buffer
}

// Returns desired size of scratch buffer in bytes.
size_t get_workspace_size(uint32_t raw_count) {
    /* TODO: your CPU code here... */
    return 0;
}

// 'launch_rle_compress'
//
// Input:
//
//   'raw_count': Number of bytes in the input buffer 'raw'.
//
//   'raw': Uncompressed bytes in GPU memory.
//
//   'workspace': Scratch buffer in GPU memory. The size of the scratch buffer
//   in bytes is determined by 'get_workspace_size'.
//
// Output:
//
//   Returns: 'compressed_count', the number of runs in the compressed data.
//
//   'compressed_data': Output buffer of size 'raw_count' in GPU memory. The
//   function should fill the first 'compressed_count' bytes of this buffer
//   with the compressed data.
//
//   'compressed_lengths': Output buffer of size 'raw_count' in GPU memory. The
//   function should fill the first 'compressed_count' integers in this buffer
//   with the lengths of the runs in the compressed data.
//
uint32_t launch_rle_compress(
    uint32_t raw_count,
    char const *raw,             // pointer to GPU buffer
    void *workspace,             // pointer to GPU buffer
    char *compressed_data,       // pointer to GPU buffer
    uint32_t *compressed_lengths // pointer to GPU buffer
) {
    using Data = typename Op::Data;
    int num_blocks =
        CEIL_DIV(n, VALS_PER_THREAD * WARPS_PER_BLOCK * 32); // 32 threads per warp
    // printf("Launching scan with %d blocks\n", num_blocks);

    Data *block_sums = reinterpret_cast<Data *>(workspace);
    Data *block_sums_workspace = block_sums + num_blocks;

    // scan each block, store block sums in workspace
    dim3 gridDim = dim3(num_blocks, 1, 1);
    dim3 blockDim = dim3(WARPS_PER_BLOCK * 32, 1, 1);
    uint32_t shmem_size_bytes = WARPS_PER_BLOCK * 32 * sizeof(Data);

    // printf("lauchining upstream \n");
    upstream_scan<Op><<<gridDim, blockDim, shmem_size_bytes>>>(n, raw, block_sums);

    if (num_blocks == 1) {
        return x;
    }

    // scan "spine" (the block sums)
    dim3 spine_gridDim(1);
    dim3 spine_blockDim(WARPS_PER_BLOCK * 32);
    uint32_t spine_shmem_size_bytes = WARPS_PER_BLOCK * 32 * sizeof(Data);
    spine_scan<Op><<<spine_gridDim, spine_blockDim, spine_shmem_size_bytes>>>(block_sums);

    // downstream fixup
    downstream_scan_fix<Op><<<gridDim, blockDim>>>(raw, block_sums);

    uint32_t compressed_count = 0;
    return compressed_count;
}

} // namespace rle_gpu

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

template <typename Reset, typename F>
double benchmark_ms(double target_time_ms, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        f();
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms);
    }
    return best_time_ms;
}

struct Results {
    double time_ms;
};

enum class Mode {
    TEST,
    BENCHMARK,
};

Results run_config(Mode mode, std::vector<char> const &raw) {
    // Allocate buffers
    size_t workspace_size = rle_gpu::get_workspace_size(raw.size());
    char *raw_gpu;
    void *workspace;
    char *compressed_data_gpu;
    uint32_t *compressed_lengths_gpu;
    CUDA_CHECK(hipMalloc(&raw_gpu, raw.size()));
    CUDA_CHECK(hipMalloc(&workspace, workspace_size));
    CUDA_CHECK(hipMalloc(&compressed_data_gpu, raw.size()));
    CUDA_CHECK(hipMalloc(&compressed_lengths_gpu, raw.size() * sizeof(uint32_t)));

    // Copy input data to GPU
    CUDA_CHECK(hipMemcpy(raw_gpu, raw.data(), raw.size(), hipMemcpyHostToDevice));

    auto reset = [&]() {
        CUDA_CHECK(hipMemset(compressed_data_gpu, 0, raw.size()));
        CUDA_CHECK(hipMemset(compressed_lengths_gpu, 0, raw.size() * sizeof(uint32_t)));
    };

    auto f = [&]() {
        rle_gpu::launch_rle_compress(
            raw.size(),
            raw_gpu,
            workspace,
            compressed_data_gpu,
            compressed_lengths_gpu);
    };

    // Test correctness
    reset();
    uint32_t compressed_count = rle_gpu::launch_rle_compress(
        raw.size(),
        raw_gpu,
        workspace,
        compressed_data_gpu,
        compressed_lengths_gpu);
    std::vector<char> compressed_data(compressed_count);
    std::vector<uint32_t> compressed_lengths(compressed_count);
    CUDA_CHECK(hipMemcpy(
        compressed_data.data(),
        compressed_data_gpu,
        compressed_count,
        hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(
        compressed_lengths.data(),
        compressed_lengths_gpu,
        compressed_count * sizeof(uint32_t),
        hipMemcpyDeviceToHost));

    std::vector<char> compressed_data_expected;
    std::vector<uint32_t> compressed_lengths_expected;
    rle_compress_cpu(
        raw.size(),
        raw.data(),
        compressed_data_expected,
        compressed_lengths_expected);

    bool correct = true;
    if (compressed_count != compressed_data_expected.size()) {
        printf("Mismatch in compressed count:\n");
        printf("  Expected: %zu\n", compressed_data_expected.size());
        printf("  Actual:   %u\n", compressed_count);
        correct = false;
    }
    if (correct) {
        for (size_t i = 0; i < compressed_data_expected.size(); i++) {
            if (compressed_data[i] != compressed_data_expected[i]) {
                printf("Mismatch in compressed data at index %zu:\n", i);
                printf(
                    "  Expected: 0x%02x\n",
                    static_cast<unsigned char>(compressed_data_expected[i]));
                printf(
                    "  Actual:   0x%02x\n",
                    static_cast<unsigned char>(compressed_data[i]));
                correct = false;
                break;
            }
            if (compressed_lengths[i] != compressed_lengths_expected[i]) {
                printf("Mismatch in compressed lengths at index %zu:\n", i);
                printf("  Expected: %u\n", compressed_lengths_expected[i]);
                printf("  Actual:   %u\n", compressed_lengths[i]);
                correct = false;
                break;
            }
        }
    }
    if (!correct) {
        if (raw.size() <= 1024) {
            printf("\nInput:\n");
            for (size_t i = 0; i < raw.size(); i++) {
                printf("  [%4zu] = 0x%02x\n", i, static_cast<unsigned char>(raw[i]));
            }
            printf("\nExpected:\n");
            for (size_t i = 0; i < compressed_data_expected.size(); i++) {
                printf(
                    "  [%4zu] = data: 0x%02x, length: %u\n",
                    i,
                    static_cast<unsigned char>(compressed_data_expected[i]),
                    compressed_lengths_expected[i]);
            }
            printf("\nActual:\n");
            if (compressed_data.size() == 0) {
                printf("  (empty)\n");
            }
            for (size_t i = 0; i < compressed_data.size(); i++) {
                printf(
                    "  [%4zu] = data: 0x%02x, length: %u\n",
                    i,
                    static_cast<unsigned char>(compressed_data[i]),
                    compressed_lengths[i]);
            }
        }
        exit(1);
    }

    if (mode == Mode::TEST) {
        return {};
    }

    // Benchmark
    double target_time_ms = 1000.0;
    double time_ms = benchmark_ms(target_time_ms, reset, f);

    // Cleanup
    CUDA_CHECK(hipFree(raw_gpu));
    CUDA_CHECK(hipFree(workspace));
    CUDA_CHECK(hipFree(compressed_data_gpu));
    CUDA_CHECK(hipFree(compressed_lengths_gpu));

    return {time_ms};
}

template <typename Rng> std::vector<char> generate_test_data(uint32_t size, Rng &rng) {
    auto random_byte = std::uniform_int_distribution<int32_t>(
        std::numeric_limits<char>::min(),
        std::numeric_limits<char>::max());
    constexpr uint32_t alphabet_size = 4;
    auto alphabet = std::vector<char>();
    for (uint32_t i = 0; i < alphabet_size; i++) {
        alphabet.push_back(random_byte(rng));
    }
    auto random_symbol = std::uniform_int_distribution<uint32_t>(0, alphabet_size - 1);
    auto data = std::vector<char>();
    for (uint32_t i = 0; i < size; i++) {
        data.push_back(alphabet.at(random_symbol(rng)));
    }
    return data;
}

int main(int argc, char const *const *argv) {
    auto rng = std::mt19937(0xCA7CAFE);

    auto test_sizes = std::vector<uint32_t>{
        16,
        10,
        128,
        100,
        1 << 10,
        1000,
        1 << 20,
        1'000'000,
        16 << 20,
    };

    printf("Correctness:\n\n");
    for (auto test_size : test_sizes) {
        auto raw = generate_test_data(test_size, rng);
        printf("  Testing compression for size %u\n", test_size);
        run_config(Mode::TEST, raw);
        printf("  OK\n\n");
    }

    auto test_data_search_paths = std::vector<std::string>{".", "/"};
    std::string test_data_path;
    for (auto test_data_search_path : test_data_search_paths) {
        auto candidate_path = test_data_search_path + "/rle_raw.bmp";
        if (std::filesystem::exists(candidate_path)) {
            test_data_path = candidate_path;
            break;
        }
    }
    if (test_data_path.empty()) {
        printf("Could not find test data file.\n");
        exit(1);
    }

    auto raw = std::vector<char>();
    {
        auto file = std::ifstream(test_data_path, std::ios::binary);
        if (!file) {
            printf("Could not open test data file '%s'.\n", test_data_path.c_str());
            exit(1);
        }
        file.seekg(0, std::ios::end);
        raw.resize(file.tellg());
        file.seekg(0, std::ios::beg);
        file.read(raw.data(), raw.size());
    }

    printf("Performance:\n\n");
    printf("  Testing compression on file 'rle_raw.bmp' (size %zu)\n", raw.size());
    auto results = run_config(Mode::BENCHMARK, raw);
    printf("  Time: %.2f ms\n", results.time_ms);

    return 0;
}